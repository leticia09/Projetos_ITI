#include "hip/hip_runtime.h"
#include <stdio.h>

__global__ void funcao1(int x) // global simboliza que essa função é uma Kernel ou seja
{                          // uma função que será executada por todas as threads de cada bloco 
                           // do grid da GPU é obrigatório o retorno VOID.
    
}

int main()
{
    int * x, * y
    funcao1<<<2,16>>>(x);

    printf("Hello world\n");
    return 0;
}