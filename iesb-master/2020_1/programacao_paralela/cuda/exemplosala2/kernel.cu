#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include <stdlib.h>
#include "omp.h"

#include <stdio.h>
#define tamanho 1000

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void preenche_vetor(unsigned long long int* vet,  unsigned long long int num);
void mostrar_vetor(unsigned long long int* vet);
void somar_vetores(unsigned long long int* a, unsigned long long int* b, unsigned long long int* c);

__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];

    

}


__global__ void somar_vetores_gpu(unsigned long long int* a, unsigned long long int* b, unsigned long long int* c)
{
    unsigned long long int qtd_threads = blockDim.x;
    unsigned long long int qtd_blocos = gridDim.x;
    unsigned long long int thread_id = threadIdx.x;
    unsigned long long int block_id = blockIdx.x;
    long int aux1, aux2;

  //  printf("\nEu sou a thread %d/%d do Bloco %d/%d", thread_id, qtd_threads-1, block_id, qtd_blocos-1);

    if (block_id < tamanho)
    {
        c[block_id] = a[block_id] + b[block_id];

        for (aux1 = 0; aux1 < 10000; aux1++)
        {
            for (aux2 = 0; aux2 < 10000; aux2++)
            {
                aux1 = aux1;
            }
        }

    }

    
}

void somar_vetores(unsigned long long int* a, unsigned long long int* b, unsigned long long int* c)
{
    int i;
    long int aux1, aux2;

    for (i = 0; i < tamanho; i++)
    {
        c[i] = a[i] + b[i];
        for (aux1 = 0; aux1 < 10000; aux1++)
        {
            for (aux2 = 0; aux2 < 10000; aux2++)
            {
                aux1 = aux1;
            }
        }
    }

}

int main()
{
    unsigned long long int* a, * b, * c, * a_gpu, * b_gpu, * c_gpu;
    unsigned long long int tam;
    tam = tamanho;
    

    double t1, t2;
        
    printf("\nTamanho = %llu", tamanho);

    t1 = omp_get_wtime();
    a = (unsigned long long int *)malloc(sizeof(unsigned long long int) * tamanho);
    b = (unsigned long long int*)malloc(sizeof(unsigned long long int) * tamanho);
    c = (unsigned long long int*)malloc(sizeof(unsigned long long int) * tamanho);
    t2 = omp_get_wtime();
    printf("\nTempo de alocacao de vetores na CPU: %lf", t2 - t1);

    t1 = omp_get_wtime();
    hipMalloc((void**)&a_gpu, sizeof(unsigned long long int) * tamanho);
    hipMalloc((void**)&b_gpu, sizeof(unsigned long long int) * tamanho);
    hipMalloc((void**)&c_gpu, sizeof(unsigned long long int) * tamanho);
    t2 = omp_get_wtime();
    printf("\nTempo de alocacao de vetores na GPU: %lf", t2 - t1);


    preenche_vetor(a, 2);
    preenche_vetor(b, 4);
    
/*
    printf("\nMostrando vetor 1: ");
    mostrar_vetor(a, tamanho);
    printf("\nMostrando vetor 2: ");
    mostrar_vetor(b, tamanho);

*/
    
    t1 = omp_get_wtime();
    hipMemcpy(a_gpu, a, tamanho * sizeof(unsigned long long int), hipMemcpyHostToDevice);
    hipMemcpy(a_gpu, b, tamanho * sizeof(unsigned long long int), hipMemcpyHostToDevice);
    t2 = omp_get_wtime();
    printf("\nTempo de copia da CPU para GPU: %lf", t2 - t1);

    t1 = omp_get_wtime();
    somar_vetores (a, b,c);
    t2 = omp_get_wtime();
    printf("\nTempo da Soma na CPU: %lf", t2 - t1);

    t1 = omp_get_wtime();
     somar_vetores_gpu << <tam, 1 >> > (a_gpu, b_gpu, c_gpu);
    t2 = omp_get_wtime();
    printf("\nTempo da Soma na GPU: %lf", t2 - t1);

    printf("\n");
    free(a);
    free(b);
    free(c);

    hipFree(a_gpu);
    hipFree(b_gpu);
    hipFree(c_gpu);
    return 0;
}


void preenche_vetor(unsigned long long int* vet, unsigned long long int num)
{
    unsigned long long int i;

    for (i = 0; i < tamanho; i++)
    {
        vet[i] = num;
    }
}


void mostrar_vetor(unsigned long long int* vet)
{
    unsigned long long int i;

    for (i = 0; i < tamanho; i++)
    {
        printf("%lld ", vet[i]);
    }
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
