#include "hip/hip_runtime.h"


#include <stdio.h>
#include "omp.h"


__global__ void funcao1(int *var_gpu)
{
    printf("a");
    *var_gpu = 2;
}

int main()
{
    int var, *var_gpu;

    var =0;
    hipMalloc((void**)&var_gpu,sizeof(int));
    hipMemcpy(var_gpu,&var,sizeof(int),hipMemcpyHostToDevice);

    funcao1<<<10,10>>>(var_gpu);
    
    hipMemcpy(&var,var_gpu,sizeof(int),hipMemcpyDeviceToHost);

    printf("var2 = %d\n",var);
    hipFree(var_gpu);
    return 0;
    
}